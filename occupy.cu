#include <hip/hip_runtime.h>
#include <algorithm>

__device__ int get_global_index(void) {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void kernel(void) {
	while(1);
}

int main(int argc, char **argv) {
	int block_size = 128;
	int grid_size = 1;
	int gpu_num;

	hipGetDeviceCount(&gpu_num);
	if (argc > 1) {
		for (int i = 1; i < argc; i++) {
			hipSetDevice(atoi(argv[i]));
			kernel<<<grid_size, block_size>>>();
		}
	}
	else {
		for (int i = 0; i < gpu_num; i++) {
			hipSetDevice(i);
			kernel<<<grid_size, block_size>>>();
		}
	}
	if (argc > 1) {
		grid_size = atoi(argv[1]);
	}
	
	hipDeviceSynchronize();

	return 0;
}
