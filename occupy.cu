#include <hip/hip_runtime.h>
#include <algorithm>

__device__ int get_global_index(void) {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void kernel(void) {
	while(1);
}

int main(int argc, char **argv) {
	int block_size = 128;
	int grid_size = 1;
	int gpu_num;

	unsigned long int bytes = 8e9; // size of memory to occupy
	float* data;

	hipGetDeviceCount(&gpu_num);
	if (argc > 1) {
		for (int i = 1; i < argc; i++) {
			hipSetDevice(atoi(argv[i]));
			hipMalloc((void**)&data, bytes);
			kernel<<<grid_size, block_size>>>();
		}
	}
	else {
		for (int i = 0; i < gpu_num; i++) {
			hipSetDevice(i);
			hipMalloc((void**)&data, bytes);
			kernel<<<grid_size, block_size>>>();
		}
	}
	
	hipDeviceSynchronize();

	return 0;
}
